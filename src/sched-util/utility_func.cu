#include "hip/hip_runtime.h"
#include "utility_func.cuh"
#include "utility_host.hpp"
#include <stdint.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <stdio.h>

// Prints a message and returns zero if the given value is not hipSuccess
#define CheckCUDAError(val) (InternalCheckCUDAError((val), #val, __FILE__, __LINE__))

// Called internally by CheckCUDAError
static inline int InternalCheckCUDAError(hipError_t result, const char *fn,
        const char *file, int line) {
    if (result == hipSuccess) return 0;
    printf("CUDA error %d in %s, line %d (%s): %s\n", (int) result, file, line,
            fn, hipGetErrorString(result));
    return -1;
}

static __global__ void getTimeInternal(uint64_t *targetTime, uint64_t startTimeOffsetNs) {
    if(threadIdx.x == 0){
        *targetTime = Util::getTimeGPU() + startTimeOffsetNs;
    }
    __syncthreads();
}

void Util::getStartTimeGPU(uint64_t *d_targetStartTime, uint64_t startTimeOffsetNs){
        getTimeInternal<<<1,1>>>(d_targetStartTime, startTimeOffsetNs);

        if (CheckCUDAError(hipDeviceSynchronize())) perror("Could not synchronize device\n");
}


int Util::getHostDeviceTimeOffset(int deviceId, uint64_t *device_ns, double *host_ns){
    uint64_t *time_d;

    if (CheckCUDAError(hipSetDevice(deviceId))) return -1;
    if (CheckCUDAError(hipMalloc((void**)&time_d, sizeof(*time_d)))) return -1;

    // Warm-up
    getTimeInternal<<<1,1>>>(time_d, 0.0);
    if (CheckCUDAError(hipDeviceSynchronize())) return -1;

    // Do Measurement
    getTimeInternal<<<1,1>>>(time_d, 0.0);
    *host_ns = Util::getCpuTimeNs();
    if (CheckCUDAError(hipMemcpy(device_ns, time_d, sizeof(*device_ns), hipMemcpyDeviceToHost))) {
        hipFree(time_d);
        return -1;
    }
    hipFree(time_d);
    return 0;
}

#define SPIN_DURATION (1000000000)

__global__ void spinKernel(uint64_t spin_duration) {
    uint64_t start_time = Util::getTimeGPU();
    while ((Util::getTimeGPU() - start_time) < spin_duration) {
        continue;
    }
}

int Util::getGpuTimeScale(int deviceId, double* scale){
    double cpuStart, cpuStop;
    if (CheckCUDAError(hipSetDevice(deviceId))) return -1;

    // Warm-up
    spinKernel<<<1,1>>>(1000);
    if (CheckCUDAError(hipDeviceSynchronize())) return -1;
    cpuStart = Util::getCpuTimeNs();
    spinKernel<<<1, 1>>>(SPIN_DURATION);
    if (CheckCUDAError(hipDeviceSynchronize())) return -1;
    cpuStop = Util::getCpuTimeNs();
    *scale = (double)(cpuStop-cpuStart)/(double)SPIN_DURATION;
    return 0;
}
